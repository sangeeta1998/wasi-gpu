#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements) {
        C[i] = A[i] + B[i];
    }
}

extern "C" void launchVectorAdd(const float *A, const float *B, float *C, int numElements) {
    float *d_A = nullptr;
    float *d_B = nullptr;
    float *d_C = nullptr;

    hipMalloc((void **)&d_A, numElements * sizeof(float));
    hipMalloc((void **)&d_B, numElements * sizeof(float));
    hipMalloc((void **)&d_C, numElements * sizeof(float));

    hipMemcpy(d_A, A, numElements * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, numElements * sizeof(float), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);

    hipMemcpy(C, d_C, numElements * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
